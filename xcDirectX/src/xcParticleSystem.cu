#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""

#include "xcParticleSystem.cuh"

namespace xcEngineSDK {

  __global__ void
    checkParticle(bool* isLiving, float* lifeTime) {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    isLiving[i] = lifeTime[i] > 0;
  }

  void
    ParticleSystem::init() {
    unsigned int threadsPerBlock = 1024;
    unsigned int blocksPerGrid = (m_maxNumberOfParticle + threadsPerBlock - 1)
      / threadsPerBlock;
  }

  void
    ParticleSystem::update(const float& deltaTime) {
  }

  void
    ParticleSystem::destroy() {
  }
}